#include "common.h"

#include <hip/hip_runtime.h>

int main() {
    hipStream_t stream0, stream1;
    void *data0, *data1;
    size_t sz = 10;

    CUDA_CALL(hipSetDevice(0));
    CUDA_CALL(hipStreamCreate(&stream0));
    CUDA_CALL(hipMalloc(&data0, sz));
    
    CUDA_CALL(hipSetDevice(1));
    CUDA_CALL(hipStreamCreate(&stream1));
    CUDA_CALL(hipMalloc(&data1, sz));

    // If the data can be copied by the target stream
    CUDA_CALL(hipMemcpyAsync(data1, data0, sz, hipMemcpyDeviceToDevice, stream1));
    CUDA_CALL(hipStreamSynchronize(stream1));

    // If the stream can be used when the device is set to another device.
    CUDA_CALL(hipSetDevice(0));
    CUDA_CALL(hipMemcpyAsync(data1, data0, sz, hipMemcpyDeviceToDevice, stream1));
    CUDA_CALL(hipStreamSynchronize(stream1));

    return 0;
}