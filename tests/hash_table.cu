#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "common.h"
#include "hash_table.cuh"


class MutableDeviceOrderedHashTable : public DeviceOrderedHashTable {
public:
 typedef typename DeviceOrderedHashTable::Bucket *Iterator;

 explicit MutableDeviceOrderedHashTable(OrderedHashTable *const hostTable)
     : DeviceOrderedHashTable(hostTable->DeviceHandle()) {}

 inline __device__ Iterator Search(const unsigned int id) {
   const unsigned int pos = SearchForPosition(id);

   return GetMutable(pos);
 }

 inline __device__ bool AttemptInsertAt(const unsigned int pos, const unsigned int id,
                                        const unsigned int index, const unsigned int version) {
   const unsigned int key = atomicCAS(&GetMutable(pos)->key, kEmptyKey, id);
   if (key == kEmptyKey || key == id) {
     atomicMin(&GetMutable(pos)->index, index);
     atomicCAS(&GetMutable(pos)->version, kEmptyKey, version);
     return true;
   } else {
     // we need to search elsewhere
     return false;
   }
 }

 inline __device__ Iterator Insert(const unsigned int id, const unsigned int index, const unsigned int version) {
   size_t pos = Hash(id);

   // linearly scan for an empty slot or matching entry
   unsigned int delta = 1;
   while (!AttemptInsertAt(pos, id, index, version)) {
     pos = Hash(pos + delta);
     delta += 1;
   }

   return GetMutable(pos);
 }

private:
 inline __device__ Iterator GetMutable(const size_t pos) {
   assert(pos < this->_size);
   // The parent class Device is read-only, but we ensure this can only be
   // constructed from a mutable version of OrderedHashTable, making this
   // a safe cast to perform.
   return const_cast<Iterator>(this->_table + pos);
 }
};

/**
* @brief Calculate the number of buckets in the hashtable. To guarantee we can
* fill the hashtable in the worst case, we must use a number of buckets which
* is a power of two.
* https://en.wikipedia.org/wiki/Quadratic_probing#Limitations
*/
size_t TableSize(const size_t num, const size_t scale) {
 const size_t next_pow2 = 1 << static_cast<size_t>(1 + std::log2(num >> 1));
 return next_pow2 << scale;
}

template <typename T>
struct BlockPrefixCallbackOp {
 T _running_total;

 __device__ BlockPrefixCallbackOp(const T running_total)
     : _running_total(running_total) {}

 __device__ T operator()(const T block_aggregate) {
   const T old_prefix = _running_total;
   _running_total += block_aggregate;
   return old_prefix;
 }
};

template <unsigned int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void
generate_hashmap_duplicates(const unsigned int *const items,
                           const size_t num_items,
                           unsigned int version,
                           MutableDeviceOrderedHashTable table) {
 assert(BLOCK_SIZE == blockDim.x);

 const size_t block_start = TILE_SIZE * blockIdx.x;
 const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
 for (size_t index = threadIdx.x + block_start; index < block_end;
      index += BLOCK_SIZE) {
   if (index < num_items) {
     printf("block %d, thread %d insert %lu with val %d\n", blockIdx.x, threadIdx.x, index, items[index]);
     table.Insert(items[index], index, version);
   }
 }
}

template <unsigned int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_hashmap_unique(const unsigned int *const items,
                                       const size_t num_items,
                                       MutableDeviceOrderedHashTable table,
                                       unsigned int global_offset,
                                       unsigned int version) {
 assert(BLOCK_SIZE == blockDim.x);

 using Iterator = typename MutableDeviceOrderedHashTable::Iterator;

 const size_t block_start = TILE_SIZE * blockIdx.x;
 const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
 for (size_t index = threadIdx.x + block_start; index < block_end; index += BLOCK_SIZE) {
   if (index < num_items) {
     const Iterator pos = table.Insert(items[index], index, version);

     // since we are only inserting unique items, we know their local id
     // will be equal to their index
     pos->local = global_offset + static_cast<unsigned int>(index);
   }
 }
}

template <unsigned int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void count_hashmap(const unsigned int *items, const size_t num_items,
                             DeviceOrderedHashTable table,
                             const unsigned int version,
                             unsigned int *const num_unique) {
 assert(BLOCK_SIZE == blockDim.x);

 using BlockReduce = typename hipcub::BlockReduce<unsigned int, BLOCK_SIZE>;
 using Bucket = typename DeviceOrderedHashTable::Bucket;

 const size_t block_start = TILE_SIZE * blockIdx.x;
 const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

 size_t count = 0;

#pragma unroll
 for (size_t index = threadIdx.x + block_start; index < block_end;
      index += BLOCK_SIZE) {
   if (index < num_items) {
     const Bucket &bucket = *table.Search(items[index]);
     printf("block %d, thread %d insert %lu with val %d, target index is %d\n", blockIdx.x, threadIdx.x, index, items[index], bucket.index);
     if (bucket.index == index && bucket.version == version) {
       ++count;
     }
   }
 }

 __shared__ typename BlockReduce::TempStorage temp_space;

 count = BlockReduce(temp_space).Sum(count);

 if (threadIdx.x == 0) {
   printf("block %d insert %lu elements\n", blockIdx.x, count);
   num_unique[blockIdx.x] = count;
   if (blockIdx.x == 0) {
     num_unique[gridDim.x] = 0;
   }
 }
}

template <unsigned int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void compact_hashmap(const unsigned int *const items,
                               const size_t num_items,
                               MutableDeviceOrderedHashTable table,
                               const unsigned int *const num_items_prefix,
                               unsigned int * const mapping,
                               size_t *const num_unique_items,
                               unsigned int global_offset,
                               unsigned int version) {
 assert(BLOCK_SIZE == blockDim.x);

 using FlagType = size_t;
 using BlockScan = typename hipcub::BlockScan<FlagType, BLOCK_SIZE>;
 using Bucket = typename DeviceOrderedHashTable::Bucket;

 constexpr size_t VALS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;

 __shared__ typename BlockScan::TempStorage temp_space;

 const size_t offset = num_items_prefix[blockIdx.x];

 BlockPrefixCallbackOp<FlagType> prefix_op(0);

 // count successful placements
 for (size_t i = 0; i < VALS_PER_THREAD; ++i) {
   const size_t index =
       threadIdx.x + i * BLOCK_SIZE + blockIdx.x * TILE_SIZE;

   FlagType flag;
   Bucket *kv;
   if (index < num_items) {
     kv = table.Search(items[index]);
     flag = kv->version == version && kv->index == index;
   } else {
     flag = 0;
   }

   if (!flag) {
       kv = nullptr;
   }

   BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
   __syncthreads();

   if (kv) {
        const unsigned int pos = global_offset + offset + flag;
        kv->local = pos;
        mapping[pos] = items[index];
        printf("Insert mapping %u with val %u\n", pos, items[index]);
   }
 }

 if (threadIdx.x == 0 && blockIdx.x == 0) {
   *num_unique_items = global_offset + num_items_prefix[gridDim.x];
 }
}

DeviceOrderedHashTable::DeviceOrderedHashTable(const Bucket *const table,
                                              const size_t size)
   : _table(table), _size(size) {}

DeviceOrderedHashTable OrderedHashTable::DeviceHandle() const {
 return DeviceOrderedHashTable(_table, _size);
}

OrderedHashTable::OrderedHashTable(const size_t size, unsigned int device, hipStream_t stream, const size_t scale)
   : _table(nullptr), _size(TableSize(size, scale)), _device(device), _global_version(0), _global_offset(0) {
 // make sure we will at least as many buckets as items.
 CUDA_CALL(hipMalloc(&_table, sizeof(Bucket) * _size));
 CUDA_CALL(hipMalloc(&_local_mapping, sizeof(unsigned int) * size));

 printf("fuck size %lu\n", size);

 CUDA_CALL(hipMemsetAsync(_table, (int)kEmptyKey,
                           sizeof(Bucket) * _size, stream));
 CUDA_CALL(hipMemsetAsync(_local_mapping, (int)kEmptyKey,
                           sizeof(unsigned int) * size, stream));
}

OrderedHashTable::~OrderedHashTable() { 
    CUDA_CALL(hipFree(_table));
    CUDA_CALL(hipFree(_local_mapping));
}

void OrderedHashTable::FillWithDuplicates(const unsigned int *const input,
                                         const size_t num_input,
                                         unsigned int *const unique,
                                         size_t &num_unique,
                                         hipStream_t stream) {
 const size_t num_tiles = (num_input + kCudaTileSize - 1) / kCudaTileSize;

 const dim3 grid(num_tiles);
 const dim3 block(kCudaBlockSize);

 auto device_table = MutableDeviceOrderedHashTable(this);

 generate_hashmap_duplicates<kCudaBlockSize, kCudaTileSize>
     <<<grid, block, 0, stream>>>(input, num_input, _global_version, device_table);
 CUDA_CALL(hipGetLastError());

 unsigned int *item_prefix;
 CUDA_CALL(hipMalloc(&item_prefix, sizeof(unsigned int) * (num_input + 1)));

 count_hashmap<kCudaBlockSize, kCudaTileSize>
     <<<grid, block, 0, stream>>>(input, num_input, device_table, _global_version, item_prefix);
 CUDA_CALL(hipGetLastError());

 size_t workspace_bytes;
 CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
     nullptr, workspace_bytes, static_cast<unsigned int *>(nullptr),
     static_cast<unsigned int *>(nullptr), grid.x + 1, stream));
 void *workspace;
 CUDA_CALL(hipMalloc(&workspace, workspace_bytes));

 CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
     workspace, workspace_bytes, item_prefix, item_prefix, grid.x + 1, stream));

 CUDA_CALL(hipFree(workspace));

 size_t *d_num_unique;
 CUDA_CALL(hipMalloc(&d_num_unique, sizeof(size_t)));
 compact_hashmap<kCudaBlockSize, kCudaTileSize><<<grid, block, 0, stream>>>(
     input, num_input, device_table, item_prefix, _local_mapping, d_num_unique, _global_offset, _global_version);
 CUDA_CALL(hipGetLastError());
 CUDA_CALL(hipFree(item_prefix));
 CUDA_CALL(hipMemcpyAsync(&num_unique, d_num_unique, sizeof(size_t), hipMemcpyDeviceToHost));
 CUDA_CALL(hipStreamSynchronize(stream));
 _global_version++;
 _global_offset = num_unique;

 CUDA_CALL(hipFree(d_num_unique));
 std::cout << num_unique << std::endl;
 CUDA_CALL(hipMemcpyAsync(unique, _local_mapping, num_unique * sizeof(unsigned int), hipMemcpyDeviceToHost, stream));
}

void OrderedHashTable::FillWithUnique(const unsigned int *const input,
                                     const size_t num_input,
                                     hipStream_t stream) {
 const size_t num_tiles = (num_input + kCudaTileSize - 1) / kCudaTileSize;

 const dim3 grid(num_tiles);
 const dim3 block(kCudaBlockSize);

 auto device_table = MutableDeviceOrderedHashTable(this);

 generate_hashmap_unique<kCudaBlockSize, kCudaTileSize>
     <<<grid, block, 0, stream>>>(input, num_input, device_table, _global_version, _global_offset);

 CUDA_CALL(hipGetLastError());

 _global_version++;
 _global_offset += num_input;
}

int main() {
    constexpr int device = 0;

    constexpr size_t num_input = 10;
    unsigned int input [num_input] = {0, 2, 4, 4, 2, 6, 101, 5, 6, 7};
    unsigned int output [num_input];
    size_t num_output;

    constexpr size_t num_input2 = 15;
    unsigned int input2 [num_input2] = {0, 1, 2, 4, 4, 2, 6, 101, 5, 6, 7, 13, 14, 15, 10};
    unsigned int output2 [num_input2];
    size_t num_output2;

    unsigned int *d_input;
    unsigned int *d_input2;

    CUDA_CALL(hipSetDevice(device));

    hipStream_t stream;
    CUDA_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    CUDA_CALL(hipMalloc(&d_input, num_input * sizeof(unsigned int)));
    CUDA_CALL(hipMalloc(&d_input2, num_input2 * sizeof(unsigned int)));

    CUDA_CALL(hipMemcpyAsync(d_input, input, num_input * sizeof(unsigned int), hipMemcpyHostToDevice, stream));
    CUDA_CALL(hipMemcpyAsync(d_input2, input2, num_input2 * sizeof(unsigned int), hipMemcpyHostToDevice, stream));

    OrderedHashTable table(num_input2, device, stream);
    table.FillWithDuplicates(d_input, num_input, output, num_output, stream);
    CUDA_CALL(hipStreamSynchronize(stream));

    std::cout << num_output << std::endl;
    for (size_t i = 0; i < num_output; i++) {
        std::cout << output[i] << ' ';
    }

    std::cout << std::endl << std::endl;

    table.FillWithDuplicates(d_input2, num_input2, output2, num_output2, stream);
    CUDA_CALL(hipStreamSynchronize(stream));

    std::cout << num_output2 << std::endl;
    for (size_t i = 0; i < num_output2; i++) {
        std::cout << output2[i] << ' ';
    }

    std::cout << std::endl;

    CUDA_CALL(hipFree(d_input));
    CUDA_CALL(hipFree(d_input2));

    CUDA_CALL(hipStreamDestroy(stream));
}
