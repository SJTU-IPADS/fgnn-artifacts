#include "hip/hip_runtime.h"
#include <cstdio>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "common.h"

__global__ void rand(unsigned long seed) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, i, 0, &state);

    size_t k = hiprand(&state) % 100;
    printf("Block %d, thread %d rand %lu\n", blockIdx.x, threadIdx.x, k);
}

int main() {
    dim3 grid(10);
    dim3 block(10);

    unsigned long seed = std::chrono::system_clock::now().time_since_epoch().count();

    CUDA_CALL(hipSetDevice(0));
    rand<<<grid, block>>>(seed);
    CUDA_CALL(hipDeviceReset());

    return 0;
}