#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <numeric>

#include "../common.h"
#include "../constant.h"
#include "../device.h"
#include "../logging.h"
#include "../run_config.h"
#include "../timer.h"
#include "cuda_random_states.h"

namespace samgraph {
namespace common {
namespace cuda {

namespace {

__global__ void init_random_states(hiprandState *states, size_t num,
                                   unsigned long seed) {
  size_t threadId = threadIdx.x + blockIdx.x * blockDim.x;
  if (threadId <= num) {
    hiprand_init(seed, threadId, 0, &states[threadId]);
  }
}

}  // namespace

GPURandomStates::GPURandomStates(SampleType sample_type,
                                 const std::vector<size_t> &fanout,
                                 const size_t batch_size, Context ctx) {
  _ctx = ctx;
  auto device = Device::Get(_ctx);

  switch (sample_type) {
    case kKHop0:
      _num_states = PredictNumNodes(batch_size, fanout, fanout.size() - 1);
      break;
    case kKHop1:
      _num_states = PredictNumNodes(batch_size, fanout, fanout.size());
      _num_states = Min(_num_states, Constant::kKHop1MaxThreads);
      break;
    case kWeightedKHop:
      _num_states = PredictNumNodes(batch_size, fanout, fanout.size());
      _num_states = Min(_num_states, Constant::kWeightedKHopMaxThreads);
      break;
    case kRandomWalk:
    default:
      CHECK(0);
  }

  _states = static_cast<hiprandState *>(
      device->AllocDataSpace(_ctx, sizeof(hiprandState) * _num_states));

  const dim3 grid(
      RoundUpDiv(_num_states, static_cast<size_t>(Constant::kCudaBlockSize)));
  const dim3 block(Constant::kCudaBlockSize);

  unsigned long seed =
      std::chrono::system_clock::now().time_since_epoch().count();
  init_random_states<<<grid, block>>>(_states, _num_states, seed);
}

GPURandomStates::~GPURandomStates() {
  auto device = Device::Get(_ctx);
  device->FreeDataSpace(_ctx, _states);
}

}  // namespace cuda
}  // namespace common
}  // namespace samgraph
