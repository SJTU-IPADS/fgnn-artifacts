#include "hip/hip_runtime.h"
#include "cuda_random_states.h"

#include <cassert>
#include <chrono>

#include "../common.h"
#include "../device.h"
#include "../logging.h"
#include "../timer.h"

namespace samgraph {
namespace common {
namespace cuda {

namespace {

__global__ void seeds_init(hiprandState *states, size_t num,
                           unsigned long seed) {
  size_t threadId = threadIdx.x + blockIdx.x * blockDim.x;
  if (threadId <= num) {
    hiprand_init(seed, threadId, 0, &states[threadId]);
  }
}

} // namespace

GPURandomStates::GPURandomStates(std::vector<int> fanouts, size_t batch_size,
                                 Context sampler_ctx) {
  Timer t1;
  auto sampler_device = Device::Get(sampler_ctx);
  // get maximum number of hiprandState usage
  long num_random_t = batch_size;
  for (auto i : fanouts) {
    num_random_t *= i;
  }
  if (num_random_t >= 0xffffffff) {
    LOG(FATAL) << "Sampling random seed size is too large";
  }

  _num_random = static_cast<size_t>(num_random_t);

  if (_num_random > maxSeedNum) {
    _num_random = maxSeedNum;
  }

  _states = static_cast<hiprandState *>(sampler_device->AllocDataSpace(
      sampler_ctx, sizeof(hiprandState) * _num_random));

  const size_t blockSize = Constant::kCudaBlockSize;
  const dim3 grid((_num_random + blockSize - 1) / blockSize);
  const dim3 block(Constant::kCudaBlockSize);

  unsigned long seed =
      std::chrono::system_clock::now().time_since_epoch().count();
  seeds_init<<<grid, block>>>(_states, _num_random, seed);

  double random_seeder_init_time = t1.Passed();
  LOG(DEBUG) << "GPURandomSeeder initialized " << _num_random
             << " seeds, random initialization coast time: "
             << random_seeder_init_time;
}

} // namespace cuda
} // namespace common
} // namespace samgraph
