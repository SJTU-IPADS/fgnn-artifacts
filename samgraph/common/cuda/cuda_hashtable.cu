#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdio>
#include <hipcub/hipcub.hpp>

#include "../common.h"
#include "../device.h"
#include "../logging.h"
#include "../timer.h"
#include "cuda_hashtable.h"
#include "cuda_utils.h"

namespace samgraph {
namespace common {
namespace cuda {

class MutableDeviceOrderedHashTable : public DeviceOrderedHashTable {
 public:
  typedef typename DeviceOrderedHashTable::BucketO2N *IteratorO2N;
  typedef typename DeviceOrderedHashTable::BucketN2O *IteratorN2O;

  explicit MutableDeviceOrderedHashTable(OrderedHashTable *const host_table)
      : DeviceOrderedHashTable(host_table->DeviceHandle()) {}

  inline __device__ IteratorO2N SearchO2N(const IdType id) {
    const IdType pos = SearchForPositionO2N(id);

    return GetMutableO2N(pos);
  }

  inline __device__ bool AttemptInsertAtO2N(const IdType pos, const IdType id,
                                            const IdType index,
                                            const IdType version) {
    const IdType key =
        atomicCAS(&GetMutableO2N(pos)->key, Constant::kEmptyKey, id);
    if (key == Constant::kEmptyKey || key == id) {
      // we either set a match key, or found a matching key, so then place the
      // minimum index in position. Match the type of atomicMin, so ignore
      // linting
      atomicMin(&GetMutableO2N(pos)->index, index);
      atomicCAS(&GetMutableO2N(pos)->version, Constant::kEmptyKey, version);
      return true;
    } else {
      // we need to search elsewhere
      return false;
    }
  }

  inline __device__ IteratorO2N InsertO2N(const IdType id, const IdType index,
                                          const IdType version) {
    IdType pos = HashO2N(id);

    // linearly scan for an empty slot or matching entry
    IdType delta = 1;
    while (!AttemptInsertAtO2N(pos, id, index, version)) {
      pos = HashO2N(pos + delta);
      delta += 1;
    }

    return GetMutableO2N(pos);
  }

  inline __device__ IteratorN2O InsertN2O(const IdType pos,
                                          const IdType global) {
    GetMutableN2O(pos)->global = global;
    return GetMutableN2O(pos);
  }

 private:
  inline __device__ IteratorO2N GetMutableO2N(const IdType pos) {
    assert(pos < this->_o2n_size);
    // The parent class Device is read-only, but we ensure this can only be
    // constructed from a mutable version of OrderedHashTable, making this
    // a safe cast to perform.
    return const_cast<IteratorO2N>(this->_o2n_table + pos);
  }

  inline __device__ IteratorN2O GetMutableN2O(const IdType pos) {
    assert(pos < this->_n2o_size);
    return const_cast<IteratorN2O>(this->_n2o_table + pos);
  }
};

/**
 * Calculate the number of buckets in the hashtable. To guarantee we can
 * fill the hashtable in the worst case, we must use a number of buckets which
 * is a power of two.
 * https://en.wikipedia.org/wiki/Quadratic_probing#Limitations
 */
size_t TableSize(const size_t num, const size_t scale) {
  const size_t next_pow2 = 1 << static_cast<size_t>(1 + std::log2(num >> 1));
  return next_pow2 << scale;
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_hashmap_duplicates(const IdType *const items,
                                            const size_t num_items,
                                            MutableDeviceOrderedHashTable table,
                                            const IdType version) {
  assert(BLOCK_SIZE == blockDim.x);

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_items) {
      table.InsertO2N(items[index], index, version);
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_hashmap_unique(const IdType *const items,
                                        const size_t num_items,
                                        MutableDeviceOrderedHashTable table,
                                        const IdType global_offset,
                                        const IdType version) {
  assert(BLOCK_SIZE == blockDim.x);

  using IteratorO2N = typename MutableDeviceOrderedHashTable::IteratorO2N;

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_items) {
      const IteratorO2N bucket = table.InsertO2N(items[index], index, version);
      IdType pos = global_offset + static_cast<IdType>(index);
      // since we are only inserting unique items, we know their local id
      // will be equal to their index
      bucket->local = pos;
      table.InsertN2O(pos, items[index]);
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void count_hashmap(const IdType *items, const size_t num_items,
                              DeviceOrderedHashTable table,
                              IdType *const num_unique, const IdType version) {
  assert(BLOCK_SIZE == blockDim.x);

  using BlockReduce = typename hipcub::BlockReduce<IdType, BLOCK_SIZE>;
  using BucketO2N = typename DeviceOrderedHashTable::BucketO2N;

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  IdType count = 0;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_items) {
      const BucketO2N &bucket = *table.SearchO2N(items[index]);
      if (bucket.index == index && bucket.version == version) {
        ++count;
      }
    }
  }

  __shared__ typename BlockReduce::TempStorage temp_space;

  count = BlockReduce(temp_space).Sum(count);

  if (threadIdx.x == 0) {
    num_unique[blockIdx.x] = count;
    if (blockIdx.x == 0) {
      num_unique[gridDim.x] = 0;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void compact_hashmap(const IdType *const items,
                                const size_t num_items,
                                MutableDeviceOrderedHashTable table,
                                const IdType *const num_items_prefix,
                                IdType *const num_unique_items,
                                const IdType global_offset,
                                const IdType version) {
  assert(BLOCK_SIZE == blockDim.x);

  using FlagType = IdType;
  using BlockScan = typename hipcub::BlockScan<FlagType, BLOCK_SIZE>;
  using BucketO2N = typename DeviceOrderedHashTable::BucketO2N;

  constexpr const IdType VALS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;

  __shared__ typename BlockScan::TempStorage temp_space;

  const IdType offset = num_items_prefix[blockIdx.x];

  BlockPrefixCallbackOp<FlagType> prefix_op(0);

  // count successful placements
  for (IdType i = 0; i < VALS_PER_THREAD; ++i) {
    const IdType index = threadIdx.x + i * BLOCK_SIZE + blockIdx.x * TILE_SIZE;

    FlagType flag;
    BucketO2N *kv;
    if (index < num_items) {
      kv = table.SearchO2N(items[index]);
      flag = kv->version == version && kv->index == index;
    } else {
      flag = 0;
    }

    if (!flag) {
      kv = nullptr;
    }

    BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
    __syncthreads();

    if (kv) {
      const IdType pos = global_offset + offset + flag;
      kv->local = pos;
      table.InsertN2O(pos, items[index]);
    }
  }

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *num_unique_items = global_offset + num_items_prefix[gridDim.x];
  }
}

// DeviceOrderedHashTable implementation
DeviceOrderedHashTable::DeviceOrderedHashTable(const BucketO2N *const o2n_table,
                                               const BucketN2O *const n2o_table,
                                               const size_t o2n_size,
                                               const size_t n2o_size)
    : _o2n_table(o2n_table),
      _n2o_table(n2o_table),
      _o2n_size(o2n_size),
      _n2o_size(n2o_size) {}

DeviceOrderedHashTable OrderedHashTable::DeviceHandle() const {
  return DeviceOrderedHashTable(_o2n_table, _n2o_table, _o2n_size, _n2o_size);
}

// OrderedHashTable implementation
OrderedHashTable::OrderedHashTable(const size_t size, Context ctx,
                                   const size_t scale)
    : _o2n_table(nullptr),
      _o2n_size(TableSize(size, scale)),
      _n2o_size(size),
      _ctx(ctx),
      _version(0),
      _num_items(0) {
  // make sure we will at least as many buckets as items.
  auto device = Device::Get(_ctx);

  _o2n_table = static_cast<BucketO2N *>(
      device->AllocDataSpace(_ctx, sizeof(BucketO2N) * _o2n_size));
  _n2o_table = static_cast<BucketN2O *>(
      device->AllocDataSpace(_ctx, sizeof(BucketN2O) * _n2o_size));

  CUDA_CALL(hipMemset(_o2n_table, (int)Constant::kEmptyKey,
                       sizeof(BucketO2N) * _o2n_size));
  CUDA_CALL(hipMemset(_n2o_table, (int)Constant::kEmptyKey,
                       sizeof(BucketN2O) * _n2o_size));
  LOG(INFO) << "cuda hashtable init with " << _o2n_size
            << " O2N table size and " << _n2o_size << " N2O table size";
}

OrderedHashTable::~OrderedHashTable() {
  Timer t;

  auto device = Device::Get(_ctx);
  device->FreeDataSpace(_ctx, _o2n_table);
  device->FreeDataSpace(_ctx, _n2o_table);

  LOG(DEBUG) << "free " << t.Passed();
}

void OrderedHashTable::Reset(StreamHandle stream) {
  auto cu_stream = static_cast<hipStream_t>(stream);
  CUDA_CALL(hipMemsetAsync(_o2n_table, (int)Constant::kEmptyKey,
                            sizeof(BucketO2N) * _o2n_size, cu_stream));
  CUDA_CALL(hipMemsetAsync(_n2o_table, (int)Constant::kEmptyKey,
                            sizeof(BucketN2O) * _n2o_size, cu_stream));
  Device::Get(_ctx)->StreamSync(_ctx, stream);
  _version = 0;
  _num_items = 0;
}

void OrderedHashTable::FillWithDuplicates(const IdType *const input,
                                          const size_t num_input,
                                          IdType *const unique,
                                          IdType *const num_unique,
                                          StreamHandle stream) {
  const size_t num_tiles = RoundUpDiv(num_input, Constant::kCudaTileSize);
  const dim3 grid(num_tiles);
  const dim3 block(Constant::kCudaBlockSize);

  auto device_table = MutableDeviceOrderedHashTable(this);
  auto device = Device::Get(_ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);

  generate_hashmap_duplicates<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(input, num_input, device_table, _version);
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "OrderedHashTable::FillWithDuplicates "
                "generate_hashmap_duplicates with "
             << num_input << " inputs";

  IdType *item_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * (grid.x + 1)));
  LOG(DEBUG) << "OrderedHashTable::FillWithDuplicates cuda item_prefix malloc "
             << ToReadableSize(sizeof(IdType) * (grid.x + 1));

  count_hashmap<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(input, num_input, device_table,
                                      item_prefix, _version);
  device->StreamSync(_ctx, stream);

  size_t workspace_bytes;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), grid.x + 1, cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace = device->AllocWorkspace(_ctx, workspace_bytes);
  LOG(DEBUG) << "OrderedHashTable::FillWithDuplicates cuda item_prefix malloc "
             << ToReadableSize(sizeof(IdType) * (num_input + 1));

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace, workspace_bytes,
                                          item_prefix, item_prefix, grid.x + 1,
                                          cu_stream));
  device->StreamSync(_ctx, stream);

  IdType *gpu_num_unique =
      static_cast<IdType *>(device->AllocWorkspace(_ctx, sizeof(IdType)));
  LOG(DEBUG)
      << "OrderedHashTable::FillWithDuplicates cuda gpu_num_unique malloc "
      << ToReadableSize(sizeof(IdType));

  compact_hashmap<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(input, num_input, device_table,
                                      item_prefix, gpu_num_unique, _num_items,
                                      _version);
  device->StreamSync(_ctx, stream);

  device->CopyDataFromTo(gpu_num_unique, 0, num_unique, 0, sizeof(IdType), _ctx,
                         CPU(), stream);
  device->StreamSync(_ctx, stream);

  // If the number of input equals to 0, the kernel won't
  // be executed then the value of num_unique will be wrong.
  // We have to manually set the num_unique on this situation.
  if (num_input == 0) {
    *num_unique = _num_items;
  }

  LOG(DEBUG) << "OrderedHashTable::FillWithDuplicates num_unique "
             << *num_unique;

  device->CopyDataFromTo(_n2o_table, 0, unique, 0,
                         sizeof(IdType) * (*num_unique), _ctx, _ctx, stream);
  device->StreamSync(_ctx, stream);

  device->FreeWorkspace(_ctx, gpu_num_unique);
  device->FreeWorkspace(_ctx, item_prefix);
  device->FreeWorkspace(_ctx, workspace);

  _version++;
  _num_items = *num_unique;
}

void OrderedHashTable::FillWithUnique(const IdType *const input,
                                      const size_t num_input,
                                      StreamHandle stream) {
  const size_t num_tiles = RoundUpDiv(num_input, Constant::kCudaTileSize);
  const dim3 grid(num_tiles);
  const dim3 block(Constant::kCudaBlockSize);

  auto device_table = MutableDeviceOrderedHashTable(this);
  auto cu_stream = static_cast<hipStream_t>(stream);

  generate_hashmap_unique<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(input, num_input, device_table,
                                      _num_items, _version);
  Device::Get(_ctx)->StreamSync(_ctx, stream);

  _version++;
  _num_items += num_input;

  LOG(DEBUG) << "OrderedHashTable::FillWithUnique insert " << num_input
             << " items, now " << _num_items << " in total";
}

}  // namespace cuda
}  // namespace common
}  // namespace samgraph