#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include "../common.h"
#include "../device.h"
#include "../logging.h"
#include "../timer.h"
#include "cuda_hashtable.h"

namespace samgraph {
namespace common {
namespace cuda {

class MutableDeviceOrderedHashTable : public DeviceOrderedHashTable {
public:
  typedef typename DeviceOrderedHashTable::BukcetO2N *Iterator0;
  typedef typename DeviceOrderedHashTable::BucketN2O *Iterator1;

  explicit MutableDeviceOrderedHashTable(OrderedHashTable *const hostTable)
      : DeviceOrderedHashTable(hostTable->DeviceHandle()) {}

  inline __device__ Iterator0 SearchO2N(const IdType id) {
    const IdType pos = SearchForPositionO2N(id);

    return GetMutableO2N(pos);
  }

  inline __device__ bool AttemptInsertAtO2N(const IdType pos, const IdType id,
                                            const IdType index,
                                            const IdType version) {
    const IdType key =
        atomicCAS(&GetMutableO2N(pos)->key, Constant::kEmptyKey, id);
    if (key == Constant::kEmptyKey || key == id) {
      // we either set a match key, or found a matching key, so then place the
      // minimum index in position. Match the type of atomicMin, so ignore
      // linting
      atomicMin(&GetMutableO2N(pos)->index, index);
      atomicCAS(&GetMutableO2N(pos)->version, Constant::kEmptyKey, version);
      return true;
    } else {
      // we need to search elsewhere
      return false;
    }
  }

  inline __device__ Iterator0 InsertO2N(const IdType id, const IdType index,
                                        const IdType version) {
    IdType pos = HashO2N(id);

    // linearly scan for an empty slot or matching entry
    IdType delta = 1;
    while (!AttemptInsertAtO2N(pos, id, index, version)) {
      pos = HashO2N(pos + delta);
      delta += 1;
    }

    return GetMutableO2N(pos);
  }

  inline __device__ Iterator1 InsertN2O(const IdType pos, const IdType global) {
    GetMutableN2O(pos)->global = global;
    return GetMutableN2O(pos);
  }

private:
  inline __device__ Iterator0 GetMutableO2N(const IdType pos) {
    assert(pos < this->_o2n_size);
    // The parent class Device is read-only, but we ensure this can only be
    // constructed from a mutable version of OrderedHashTable, making this
    // a safe cast to perform.
    return const_cast<Iterator0>(this->_o2n_table + pos);
  }

  inline __device__ Iterator1 GetMutableN2O(const IdType pos) {
    assert(pos < this->_n2o_size);
    return const_cast<Iterator1>(this->_n2o_table + pos);
  }
};

/**
 * Calculate the number of buckets in the hashtable. To guarantee we can
 * fill the hashtable in the worst case, we must use a number of buckets which
 * is a power of two.
 * https://en.wikipedia.org/wiki/Quadratic_probing#Limitations
 */
size_t TableSize(const size_t num, const size_t scale) {
  const size_t next_pow2 = 1 << static_cast<size_t>(1 + std::log2(num >> 1));
  return next_pow2 << scale;
}

/**
 * This structure is used with cub's block-level prefixscan in order to
 * keep a running sum as items are iteratively processed.
 */
template <typename T> struct BlockPrefixCallbackOp {
  T _running_total;

  __device__ BlockPrefixCallbackOp(const T running_total)
      : _running_total(running_total) {}

  __device__ T operator()(const T block_aggregate) {
    const T old_prefix = _running_total;
    _running_total += block_aggregate;
    return old_prefix;
  }
};

template <int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_hashmap_duplicates(const IdType *const items,
                                            const size_t num_items,
                                            MutableDeviceOrderedHashTable table,
                                            const IdType version) {
  assert(BLOCK_SIZE == blockDim.x);

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_items) {
      table.InsertO2N(items[index], index, version);
    }
  }
}

template <int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void
generate_hashmap_unique(const IdType *const items, const size_t num_items,
                        MutableDeviceOrderedHashTable table,
                        const IdType global_offset, const IdType version) {
  assert(BLOCK_SIZE == blockDim.x);

  using Iterator0 = typename MutableDeviceOrderedHashTable::Iterator0;

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_items) {
      const Iterator0 bucket = table.InsertO2N(items[index], index, version);
      IdType pos = global_offset + static_cast<IdType>(index);
      // since we are only inserting unique items, we know their local id
      // will be equal to their index
      bucket->local = pos;
      table.InsertN2O(pos, items[index]);
    }
  }
}

template <int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void count_hashmap(const IdType *items, const size_t num_items,
                              DeviceOrderedHashTable table,
                              IdType *const num_unique, const IdType version) {
  assert(BLOCK_SIZE == blockDim.x);

  using BlockReduce = typename hipcub::BlockReduce<IdType, BLOCK_SIZE>;
  using BukcetO2N = typename DeviceOrderedHashTable::BukcetO2N;

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  IdType count = 0;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_items) {
      const BukcetO2N &bucket = *table.SearchO2N(items[index]);
      if (bucket.index == index && bucket.version == version) {
        ++count;
      }
    }
  }

  __shared__ typename BlockReduce::TempStorage temp_space;

  count = BlockReduce(temp_space).Sum(count);

  if (threadIdx.x == 0) {
    num_unique[blockIdx.x] = count;
    if (blockIdx.x == 0) {
      num_unique[gridDim.x] = 0;
    }
  }
}

template <int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void
compact_hashmap(const IdType *const items, const size_t num_items,
                MutableDeviceOrderedHashTable table,
                const IdType *const num_items_prefix,
                size_t *const num_unique_items, const IdType global_offset,
                const IdType version) {
  assert(BLOCK_SIZE == blockDim.x);

  using FlagType = IdType;
  using BlockScan = typename hipcub::BlockScan<FlagType, BLOCK_SIZE>;
  using BukcetO2N = typename DeviceOrderedHashTable::BukcetO2N;

  constexpr const int32_t VALS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;

  __shared__ typename BlockScan::TempStorage temp_space;

  const IdType offset = num_items_prefix[blockIdx.x];

  BlockPrefixCallbackOp<FlagType> prefix_op(0);

  // count successful placements
  for (int32_t i = 0; i < VALS_PER_THREAD; ++i) {
    const IdType index = threadIdx.x + i * BLOCK_SIZE + blockIdx.x * TILE_SIZE;

    FlagType flag;
    BukcetO2N *kv;
    if (index < num_items) {
      kv = table.SearchO2N(items[index]);
      flag = kv->version == version && kv->index == index;
    } else {
      flag = 0;
    }

    if (!flag) {
      kv = nullptr;
    }

    BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
    __syncthreads();

    if (kv) {
      const IdType pos = global_offset + offset + flag;
      kv->local = pos;
      table.InsertN2O(pos, items[index]);
    }
  }

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *num_unique_items = global_offset + num_items_prefix[gridDim.x];
  }
}

// DeviceOrderedHashTable implementation
DeviceOrderedHashTable::DeviceOrderedHashTable(const BukcetO2N *const o2n_table,
                                               const BucketN2O *const n2o_table,
                                               const size_t o2n_size,
                                               const size_t n2o_size)
    : _o2n_table(o2n_table), _n2o_table(n2o_table), _o2n_size(o2n_size),
      _n2o_size(n2o_size) {}

DeviceOrderedHashTable OrderedHashTable::DeviceHandle() const {
  return DeviceOrderedHashTable(_o2n_table, _n2o_table, _o2n_size, _n2o_size);
}

// OrderedHashTable implementation
OrderedHashTable::OrderedHashTable(const size_t size, Context ctx,
                                   StreamHandle stream, const size_t scale)
    : _o2n_table(nullptr), _o2n_size(TableSize(size, scale)), _n2o_size(size),
      _ctx(ctx), _version(0), _num_items(0) {
  // make sure we will at least as many buckets as items.
  auto device = Device::Get(_ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);

  _o2n_table = static_cast<BukcetO2N *>(
      device->AllocDataSpace(_ctx, sizeof(BukcetO2N) * _o2n_size));
  _n2o_table = static_cast<BucketN2O *>(
      device->AllocDataSpace(_ctx, sizeof(BucketN2O) * _n2o_size));

  CUDA_CALL(hipMemsetAsync(_o2n_table, (int)Constant::kEmptyKey,
                            sizeof(BukcetO2N) * _o2n_size, cu_stream));
  CUDA_CALL(hipMemsetAsync(_n2o_table, (int)Constant::kEmptyKey,
                            sizeof(BucketN2O) * _n2o_size, cu_stream));
  device->StreamSync(_ctx, stream);
}

OrderedHashTable::~OrderedHashTable() {
  Timer t;

  auto device = Device::Get(_ctx);
  device->FreeDataSpace(_ctx, _o2n_table);
  device->FreeDataSpace(_ctx, _n2o_table);

  LOG(DEBUG) << "free " << t.Passed();
}

void OrderedHashTable::Reset(StreamHandle stream) {
  auto cu_stream = static_cast<hipStream_t>(stream);
  CUDA_CALL(hipMemsetAsync(_o2n_table, (int)Constant::kEmptyKey,
                            sizeof(BukcetO2N) * _o2n_size, cu_stream));
  CUDA_CALL(hipMemsetAsync(_n2o_table, (int)Constant::kEmptyKey,
                            sizeof(BucketN2O) * _n2o_size, cu_stream));
  Device::Get(_ctx)->StreamSync(_ctx, stream);
  _version = 0;
  _num_items = 0;
}

void OrderedHashTable::FillWithDuplicates(const IdType *const input,
                                          const size_t num_input,
                                          IdType *const unique,
                                          size_t *const num_unique,
                                          StreamHandle stream) {
  const size_t num_tiles =
      (num_input + Constant::kCudaTileSize - 1) / Constant::kCudaTileSize;

  const dim3 grid(num_tiles);
  const dim3 block(Constant::kCudaBlockSize);

  auto device_table = MutableDeviceOrderedHashTable(this);
  auto device = Device::Get(_ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);

  generate_hashmap_duplicates<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(input, num_input, device_table, _version);
  device->StreamSync(_ctx, stream);

  IdType *item_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * (grid.x + 1)));
  LOG(DEBUG) << "OrderedHashTable::FillWithDuplicates cuda item_prefix malloc "
             << ToReadableSize(sizeof(IdType) * (grid.x + 1));

  count_hashmap<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(input, num_input, device_table,
                                      item_prefix, _version);
  device->StreamSync(_ctx, stream);

  size_t workspace_bytes;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), grid.x + 1, cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace = device->AllocWorkspace(_ctx, workspace_bytes);
  LOG(DEBUG) << "OrderedHashTable::FillWithDuplicates cuda item_prefix malloc "
             << ToReadableSize(sizeof(IdType) * (num_input + 1));

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace, workspace_bytes,
                                          item_prefix, item_prefix, grid.x + 1,
                                          cu_stream));
  device->StreamSync(_ctx, stream);

  size_t *gpu_num_unique =
      static_cast<size_t *>(device->AllocWorkspace(_ctx, sizeof(size_t)));
  LOG(DEBUG)
      << "OrderedHashTable::FillWithDuplicates cuda gpu_num_unique malloc "
      << ToReadableSize(sizeof(size_t));

  compact_hashmap<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(input, num_input, device_table,
                                      item_prefix, gpu_num_unique, _num_items,
                                      _version);
  device->StreamSync(_ctx, stream);

  device->CopyDataFromTo(gpu_num_unique, 0, num_unique, 0, sizeof(size_t), _ctx,
                         CPU(), stream);
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "OrderedHashTable::FillWithDuplicates num_unique "
             << *num_unique;

  device->CopyDataFromTo(_n2o_table, 0, unique, 0,
                         sizeof(IdType) * (*num_unique), _ctx, _ctx, stream);
  device->StreamSync(_ctx, stream);

  device->FreeWorkspace(_ctx, gpu_num_unique);
  device->FreeWorkspace(_ctx, item_prefix);
  device->FreeWorkspace(_ctx, workspace);

  _version++;
  _num_items = *num_unique;
}

void OrderedHashTable::FillWithUnique(const IdType *const input,
                                      const size_t num_input,
                                      StreamHandle stream) {

  const size_t num_tiles =
      (num_input + Constant::kCudaTileSize - 1) / Constant::kCudaTileSize;

  const dim3 grid(num_tiles);
  const dim3 block(Constant::kCudaBlockSize);

  auto device_table = MutableDeviceOrderedHashTable(this);
  auto cu_stream = static_cast<hipStream_t>(stream);

  generate_hashmap_unique<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(input, num_input, device_table,
                                      _num_items, _version);
  Device::Get(_ctx)->StreamSync(_ctx, stream);

  _version++;
  _num_items += num_input;

  LOG(DEBUG) << "OrderedHashTable::FillWithUnique insert " << num_input
             << " items, now " << _num_items << " in total";
}

} // namespace cuda
} // namespace common
} // namespace samgraph