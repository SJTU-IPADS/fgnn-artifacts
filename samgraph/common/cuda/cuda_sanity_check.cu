#include "hip/hip_runtime.h"
#include "../constant.h"
#include "../device.h"
#include "cuda_function.h"

namespace samgraph {
namespace common {
namespace cuda {

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void batch_sanity_check(IdType *map, const IdType *input,
                                   const size_t num_input) {
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_input) {
      if (map[input[index]] > 0) {
        printf("duplicate batch input");
      }
      assert(map[input[index]] == 0);
      map[input[index]] = 1;
    }
  }
}

void GPUBatchSanityCheck(IdType *map, const IdType *input,
                         const size_t num_input, Context ctx,
                         StreamHandle stream) {
  auto device = Device::Get(ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);

  const uint32_t num_tiles =
      (num_input + Constant::kCudaTileSize - 1) / Constant::kCudaTileSize;

  const dim3 grid(num_tiles);
  const dim3 block(Constant::kCudaBlockSize);

  batch_sanity_check<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(map, input, num_input);

  device->StreamSync(ctx, stream);
}

} // namespace cuda
} // namespace common
} // namespace samgraph
