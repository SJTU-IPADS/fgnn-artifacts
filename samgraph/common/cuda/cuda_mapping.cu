#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>

#include "../constant.h"
#include "../device.h"
#include "cuda_function.h"
#include "cuda_hashtable.h"

namespace samgraph {
namespace common {
namespace cuda {

template <int BLOCK_SIZE, size_t TILE_SIZE>
__device__ void map_node_ids(const IdType *const global,
                             IdType *const new_global, const size_t num_input,
                             const DeviceOrderedHashTable &table) {
  assert(BLOCK_SIZE == blockDim.x);

  using Bucket = typename OrderedHashTable::BucketO2N;

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = min(TILE_SIZE * (blockIdx.x + 1), num_input);

  for (size_t idx = threadIdx.x + block_start; idx < block_end;
       idx += BLOCK_SIZE) {
    const Bucket &bucket = *table.SearchO2N(global[idx]);
    new_global[idx] = bucket.local;
  }
}

template <int BLOCK_SIZE, size_t TILE_SIZE>
__global__ void
map_edge_ids(const IdType *const global_src, IdType *const new_global_src,
             const IdType *const global_dst, IdType *const new_global_dst,
             const size_t num_edges, DeviceOrderedHashTable table) {
  assert(BLOCK_SIZE == blockDim.x);
  assert(2 == gridDim.y);

  if (blockIdx.y == 0) {
    map_node_ids<BLOCK_SIZE, TILE_SIZE>(global_src, new_global_src, num_edges,
                                        table);
  } else {
    map_node_ids<BLOCK_SIZE, TILE_SIZE>(global_dst, new_global_dst, num_edges,
                                        table);
  }
}

void GPUMapEdges(const IdType *const global_src, IdType *const new_global_src,
                 const IdType *const global_dst, IdType *const new_global_dst,
                 const size_t num_edges, DeviceOrderedHashTable table,
                 Context ctx, StreamHandle stream) {
  const size_t num_tiles =
      (num_edges + Constant::kCudaTileSize - 1) / Constant::kCudaTileSize;
  const dim3 grid(num_tiles, 2);
  const dim3 block(Constant::kCudaBlockSize);
  auto cu_stream = static_cast<hipStream_t>(stream);

  map_edge_ids<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(global_src, new_global_src, global_dst,
                                      new_global_dst, num_edges, table);
  Device::Get(ctx)->StreamSync(ctx, stream);
}

} // namespace cuda
} // namespace common
} // namespace samgraph
