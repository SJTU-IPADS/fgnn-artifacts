#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>

#include "../constant.h"
#include "../device.h"
#include "../logging.h"
#include "cuda_frequency_hashmap.h"
#include "cuda_utils.h"

namespace samgraph {
namespace common {
namespace cuda {

namespace {

size_t TableSize(const size_t num, const size_t scale) {
  const size_t next_pow2 = 1 << static_cast<size_t>(1 + std::log2(num >> 1));
  return next_pow2 << scale;
}

class MutableDeviceFrequencyHashmap : public DeviceFrequencyHashmap {
 public:
  typedef typename DeviceFrequencyHashmap::NodeBucket *NodeIterator;
  typedef typename DeviceFrequencyHashmap::EdgeBucket *EdgeIterator;

  explicit MutableDeviceFrequencyHashmap(FrequencyHashmap *const host_map)
      : DeviceFrequencyHashmap(host_map->DeviceHandle()) {}

  inline __device__ NodeIterator SearchNode(const IdType id) {
    const IdType pos = SearchNodeForPosition(id);
    return GetMutableNode(pos);
  }

  inline __device__ EdgeIterator SearchEdge(const IdType src,
                                            const IdType dst) {
    const LongIdType pos = SearchEdgeForPosition(src, dst);
    return GetMutableEdge(pos);
  }

  inline __device__ bool AttemptInsertNodeAt(const IdType pos,
                                             const IdType id) {
    NodeIterator bucket = GetMutableNode(pos);
    const IdType key = atomicCAS(&bucket->key, Constant::kEmptyKey, id);
    if (key == Constant::kEmptyKey || key == id) {
      return true;
    } else {
      return false;
    }
  }

  inline __device__ NodeIterator InsertNode(const IdType id) {
    IdType pos = NodeHash(id);

    IdType delta = 1;
    while (!AttemptInsertNodeAt(pos, id)) {
      pos = NodeHash(pos + delta);
      delta += 1;
    }
    return GetMutableNode(pos);
  }

  inline __device__ bool AttemptInsertEdgeAt(const LongIdType pos,
                                             const LongIdType edge_id,
                                             const IdType node_id,
                                             const IdType index) {
    EdgeIterator edge_iter = GetMutableEdge(pos);
    const LongIdType key =
        atomicCAS(&edge_iter->key, Constant::kEmptyLongKey, edge_id);
    if (key == Constant::kEmptyLongKey || key == edge_id) {
      atomicAdd(&edge_iter->count, 1U);
      atomicMin(&edge_iter->index, index);
      if (key == Constant::kEmptyLongKey) {
        NodeIterator node_iter = SearchNode(node_id);
        atomicAdd(&node_iter->count, 1U);
      }
      return true;
    } else {
      return false;
    }
  }

  inline __device__ EdgeIterator InsertEdge(const IdType src, const IdType dst,
                                            const IdType index) {
    LongIdType id = EncodeEdge(src, dst);
    LongIdType pos = EdgeHash(id);

    LongIdType delta = 1;
    while (!AttemptInsertEdgeAt(pos, id, src, index)) {
      pos = EdgeHash(pos + delta);
      delta += 1;
    }

    return GetMutableEdge(pos);
  }

  inline __device__ NodeIterator GetMutableNode(const IdType pos) {
    assert(pos < _ntable_size);
    return const_cast<NodeIterator>(_node_table + pos);
  }

  inline __device__ EdgeIterator GetMutableEdge(const LongIdType pos) {
    assert(pos < _etable_size);
    return const_cast<EdgeIterator>(_edge_table + pos);
  }
};

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void init_node_table(MutableDeviceFrequencyHashmap table,
                                const size_t num_bucket) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using NodeIterator = typename MutableDeviceFrequencyHashmap::NodeIterator;

#pragma unroll
  for (IdType index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_bucket) {
      NodeIterator node_iter = table.GetMutableNode(index);
      node_iter->key = Constant::kEmptyKey;
      node_iter->count = 0;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void init_edge_table(MutableDeviceFrequencyHashmap table,
                                const size_t num_bucket) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using EdgeIterator = typename MutableDeviceFrequencyHashmap::EdgeIterator;

#pragma unroll
  for (LongIdType index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_bucket) {
      EdgeIterator edge_iter = table.GetMutableEdge(index);
      edge_iter->key = Constant::kEmptyLongKey;
      edge_iter->count = 0;
      edge_iter->index = Constant::kEmptyKey;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void reset_node_table(MutableDeviceFrequencyHashmap table,
                                 IdType *nodes, const size_t num_nodes) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using NodeIterator = typename MutableDeviceFrequencyHashmap::NodeIterator;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_nodes) {
      IdType id = nodes[index];
      IdType pos = table.SearchNodeForPosition(id);
      NodeIterator node_iter = table.GetMutableNode(pos);
      node_iter->key = Constant::kEmptyKey;
      node_iter->count = 0;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void reset_edge_table(MutableDeviceFrequencyHashmap table,
                                 IdType *unique_src, IdType *unique_dst,
                                 const size_t num_unique) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using EdgeIterator = typename MutableDeviceFrequencyHashmap::EdgeIterator;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_unique) {
      IdType src = unique_src[index];
      IdType dst = unique_dst[index];
      LongIdType pos = table.SearchEdgeForPosition(src, dst);
      printf("Search edge src %u dst %u \n", src, dst);
      EdgeIterator edge_iter = table.GetMutableEdge(pos);
      edge_iter->key = Constant::kEmptyLongKey;
      edge_iter->count = 0;
      edge_iter->index = Constant::kEmptyKey;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void populate_node_table(const IdType *nodes,
                                    const size_t num_input_node,
                                    MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_input_node) {
      table.InsertNode(nodes[index]);
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void count_frequency(const IdType *input_src,
                                const IdType *input_dst,
                                const size_t num_input_edge,
                                MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_input_edge && input_src[index] != Constant::kEmptyKey) {
      table.InsertEdge(input_src[index], input_dst[index], index);
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void count_unique_edges(const IdType *input_src,
                                   const IdType *input_dst,
                                   const size_t num_input_edge,
                                   IdType *item_prefix,
                                   DeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);

  using BlockReduce = typename hipcub::BlockReduce<IdType, BLOCK_SIZE>;
  using EdgeBucket = typename DeviceFrequencyHashmap::EdgeBucket;

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  IdType count = 0;
#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_input_edge && input_src[index] != Constant::kEmptyKey) {
      const EdgeBucket &bucket =
          *table.SearchEdge(input_src[index], input_dst[index]);
      if (bucket.index == index) {
        ++count;
      }
    }
  }

  __shared__ typename BlockReduce::TempStorage temp_space;

  count = BlockReduce(temp_space).Sum(count);

  if (threadIdx.x == 0) {
    item_prefix[blockIdx.x] = count;
    if (blockIdx.x == 0) {
      item_prefix[gridDim.x] = 0;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_unique_edges(const IdType *input_src,
                                      const IdType *input_dst,
                                      const size_t num_input_edge,
                                      IdType *item_prefix, IdType *unique_src,
                                      IdType *unique_dst, size_t *num_unique,
                                      MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);

  using FlagType = IdType;
  using BlockScan = typename hipcub::BlockScan<FlagType, BLOCK_SIZE>;
  using EdgeBucket = typename DeviceFrequencyHashmap::EdgeBucket;

  constexpr const IdType VALS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;

  __shared__ typename BlockScan::TempStorage temp_space;

  const IdType offset = item_prefix[blockIdx.x];

  BlockPrefixCallbackOp<FlagType> prefix_op(0);

  // count successful placements
  for (IdType i = 0; i < VALS_PER_THREAD; ++i) {
    const IdType index = threadIdx.x + i * BLOCK_SIZE + blockIdx.x * TILE_SIZE;

    FlagType flag;
    EdgeBucket *bucket;
    if (index < num_input_edge && input_src[index] != Constant::kEmptyKey) {
      bucket = table.SearchEdge(input_src[index], input_dst[index]);
      flag = bucket->index == index;
    } else {
      flag = 0;
    }

    if (!flag) {
      bucket = nullptr;
    }

    BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
    __syncthreads();

    if (bucket) {
      const IdType pos = offset + flag;
      unique_src[pos] = input_src[index];
      unique_dst[pos] = input_dst[index];
    }
  }

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *num_unique = item_prefix[gridDim.x];
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_unique_edge_frequency(const IdType *unique_src,
                                               const IdType *unique_dst,
                                               IdType *unique_count,
                                               const size_t num_unique,
                                               DeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using EdgeBucket = typename DeviceFrequencyHashmap::EdgeBucket;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_unique) {
      const EdgeBucket &bucket =
          *table.SearchEdge(unique_src[index], unique_dst[index]);
      unique_count[index] = bucket.count;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_num_edge(const IdType *nodes, const size_t num_nodes,
                                  const size_t K, IdType *num_edge_prefix,
                                  IdType *num_output_prefix,
                                  DeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using NodeBucket = typename DeviceFrequencyHashmap::NodeBucket;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_nodes) {
      const NodeBucket &bucket = *table.SearchNode(nodes[index]);
      num_edge_prefix[index] = bucket.count;
      num_output_prefix[index] = bucket.count > K ? K : bucket.count;
    }
  }

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    num_edge_prefix[num_nodes] = 0;
    num_output_prefix[num_nodes] = 0;
  }
}

__global__ void compact_output(const IdType *unique_src,
                               const IdType *unique_dst,
                               const IdType *unique_frequency,
                               const size_t num_nodes, const size_t K,
                               const IdType *num_unique_prefix,
                               const IdType *num_output_prefix,
                               IdType *output_src, IdType *output_dst,
                               IdType *output_data, size_t *num_output) {
  size_t i = blockIdx.x * blockDim.y + threadIdx.y;
  const size_t stride = blockDim.y * gridDim.x;

  while (i < num_nodes) {
    IdType k = threadIdx.x;
    IdType max_output = num_output_prefix[i + 1] - num_output_prefix[i];
    while (k < K && k < max_output) {
      IdType from_off = num_unique_prefix[i] + k;
      IdType to_off = num_output_prefix[i] + k;

      output_src[to_off] = unique_src[from_off];
      output_dst[to_off] = unique_dst[from_off];
      output_data[to_off] = unique_frequency[from_off];

      k += blockDim.x;
    }

    i += stride;
  }

  if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
    *num_output = num_output_prefix[num_nodes];
  }
}

}  // namespace

DeviceFrequencyHashmap::DeviceFrequencyHashmap(
    const NodeBucket *node_table, const EdgeBucket *edge_table,
    const size_t ntable_size, const size_t etable_size,
    const IdType *unique_src, const IdType *unique_dst,
    const IdType *unique_count, const size_t unique_size)
    : _node_table(node_table),
      _edge_table(edge_table),
      _ntable_size(ntable_size),
      _etable_size(etable_size),
      _unique_src(unique_src),
      _unique_dst(unique_dst),
      _unique_count(unique_count),
      _unique_size(unique_size) {}

DeviceFrequencyHashmap FrequencyHashmap::DeviceHandle() const {
  return DeviceFrequencyHashmap(_node_table, _edge_table, _ntable_size,
                                _etable_size, _unique_src, _unique_dst,
                                _unique_frequency, _num_unique);
}

FrequencyHashmap::FrequencyHashmap(const size_t max_nodes,
                                   const size_t max_edges, Context ctx,
                                   const size_t scale)
    : _ctx(ctx),
      _ntable_size(TableSize(max_nodes, scale)),
      _etable_size(TableSize(max_edges, scale)),
      _num_node(0),
      _node_list_size(max_nodes),
      _num_unique(0),
      _unique_list_size(max_edges) {
  auto device = Device::Get(_ctx);
  CHECK_EQ(_ctx.device_type, kGPU);

  _node_table = static_cast<NodeBucket *>(
      device->AllocDataSpace(_ctx, sizeof(NodeBucket) * _ntable_size));
  _edge_table = static_cast<EdgeBucket *>(
      device->AllocDataSpace(_ctx, sizeof(EdgeBucket) * _etable_size));

  _node_list = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _node_list_size));

  _unique_src = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size));
  _unique_dst = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size));
  _unique_frequency = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size));

  auto device_table = MutableDeviceFrequencyHashmap(this);
  dim3 grid0(RoundUpDiv(_node_list_size, Constant::kCudaTileSize));
  dim3 grid1(RoundUpDiv(_unique_list_size, Constant::kCudaTileSize));
  dim3 block0(Constant::kCudaBlockSize);
  dim3 block1(Constant::kCudaBlockSize);

  init_node_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid0, block0>>>(device_table, _node_list_size);
  init_edge_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid1, block1>>>(device_table, _unique_list_size);
}

FrequencyHashmap::~FrequencyHashmap() {
  auto device = Device::Get(_ctx);

  device->FreeDataSpace(_ctx, _node_table);
  device->FreeDataSpace(_ctx, _edge_table);
  device->FreeDataSpace(_ctx, _node_list);
  device->FreeDataSpace(_ctx, _unique_src);
  device->FreeDataSpace(_ctx, _unique_dst);
  device->FreeDataSpace(_ctx, _unique_frequency);
}

void FrequencyHashmap::GetTopK(const IdType *input_src, const IdType *input_dst,
                               const size_t num_input_edge,
                               const IdType *input_nodes,
                               const size_t num_input_node, const size_t K,
                               IdType *output_src, IdType *output_dst,
                               IdType *output_data, size_t *num_output,
                               StreamHandle stream) {
  const size_t num_tiles0 = RoundUpDiv(num_input_edge, Constant::kCudaTileSize);
  const size_t num_tiles1 = RoundUpDiv(num_input_node, Constant::kCudaTileSize);
  const dim3 grid0(num_tiles0);
  const dim3 grid1(num_tiles1);

  const dim3 block0(Constant::kCudaBlockSize);
  const dim3 block1(Constant::kCudaBlockSize);

  dim3 block2(Constant::kCudaBlockSize, 1);
  while (static_cast<size_t>(block2.x) >= 2 * K) {
    block2.x /= 2;
    block2.y *= 2;
  }
  dim3 grid2(RoundUpDiv(num_input_node, static_cast<size_t>(block2.y)));

  auto device_table = MutableDeviceFrequencyHashmap(this);
  auto device = Device::Get(_ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);

  size_t workspace_bytes0;
  size_t workspace_bytes1;
  size_t workspace_bytes2;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes0, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), grid0.x + 1, cu_stream));
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes1, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), grid1.x + 1, cu_stream));
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes2, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), num_input_node + 1, cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace0 = device->AllocWorkspace(_ctx, workspace_bytes0);
  void *workspace1 = device->AllocWorkspace(_ctx, workspace_bytes1);
  void *workspace2 = device->AllocWorkspace(_ctx, workspace_bytes2);

  // 0. populate the node table
  populate_node_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid1, block1, 0, cu_stream>>>(input_nodes, num_input_node,
                                        device_table);
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 0 finish";

  // 1. count frequency of every unique edge and
  //    count unique edges for every node
  count_frequency<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid0, block0, 0, cu_stream>>>(input_src, input_dst, num_input_edge,
                                        device_table);
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 1 finish";

  // 2. count the number of unique edges.
  //    prefix sum the the array
  IdType *num_unique_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * (grid0.x + 1)));
  count_unique_edges<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid0, block0, 0, cu_stream>>>(input_src, input_dst, num_input_edge,
                                        num_unique_prefix, device_table);
  device->StreamSync(_ctx, stream);

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace0, workspace_bytes0,
                                          num_unique_prefix, num_unique_prefix,
                                          grid0.x + 1, cu_stream));
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 2 finish";

  // 3. get the array of all unique edges.
  size_t *device_num_unique =
      static_cast<size_t *>(device->AllocWorkspace(_ctx, sizeof(size_t)));
  generate_unique_edges<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid0, block0, 0, cu_stream>>>(
          input_src, input_dst, num_input_edge, num_unique_prefix, _unique_src,
          _unique_dst, device_num_unique, device_table);
  device->StreamSync(_ctx, stream);

  device->CopyDataFromTo(device_num_unique, 0, &_num_unique, 0, sizeof(size_t),
                         _ctx, CPU(), stream);
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 3 finish with number of unique "
             << _num_unique;

  // 4. pair-sort unique edges(src, dst) using src as key.
  size_t workspace_bytes3;
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
      nullptr, workspace_bytes3, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), _num_unique, 0, sizeof(IdType) * 8,
      cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace3 = device->AllocWorkspace(_ctx, workspace_bytes3);
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
      workspace3, workspace_bytes3, _unique_src, _unique_src, _unique_dst,
      _unique_dst, _num_unique, 0, sizeof(IdType) * 8, cu_stream));
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 4 finish";

  // 5. get the frequency array in the order of sorted edge.
  const size_t num_tiles2 = RoundUpDiv(_num_unique, Constant::kCudaTileSize);
  const dim3 grid3(num_tiles2);
  const dim3 block3(Constant::kCudaBlockSize);
  generate_unique_edge_frequency<Constant::kCudaBlockSize,
                                 Constant::kCudaTileSize>
      <<<grid3, block3, 0, cu_stream>>>(_unique_src, _unique_dst,
                                        _unique_frequency, _num_unique,
                                        device_table);
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 5 finish";

  // 6. sort the unique src node array.
  device->CopyDataFromTo(input_nodes, 0, _node_list, 0,
                         num_input_node * sizeof(IdType), _ctx, _ctx, stream);
  device->StreamSync(_ctx, stream);
  _num_node = num_input_node;

  size_t workspace_bytes4;
  CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(
      nullptr, workspace_bytes4, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), num_input_node, 0, sizeof(IdType) * 8,
      cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace4 = device->AllocWorkspace(_ctx, workspace_bytes4);
  CUDA_CALL(hipcub::DeviceRadixSort::SortKeys(
      workspace4, workspace_bytes4, _node_list, _node_list, num_input_node, 0,
      sizeof(IdType) * 8, cu_stream));
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 6 finish";

  // 7. get array unique edge number in the order of src nodes.
  //    also count the number of output edges for each nodes.
  //    prefix sum for array of unique edge number.
  IdType *num_edge_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, (num_input_node + 1) * sizeof(IdType)));
  IdType *num_output_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, (num_input_node + 1) * sizeof(IdType)));
  generate_num_edge<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid1, block1, 0, cu_stream>>>(_node_list, num_input_node, K,
                                        num_edge_prefix, num_output_prefix,
                                        device_table);
  device->StreamSync(_ctx, stream);

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace2, workspace_bytes2,
                                          num_edge_prefix, num_edge_prefix,
                                          num_input_node + 1, cu_stream));
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 7 finish";

  // 8. segment-sort the edge for every node using the frequency as key
  //    and the dst as value.
  size_t workspace_bytes5;
  CUDA_CALL(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      nullptr, workspace_bytes5, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), _num_unique, num_input_node,
      static_cast<IdType *>(nullptr), static_cast<IdType *>(nullptr), 0,
      sizeof(IdType) * 8, cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace5 = device->AllocWorkspace(_ctx, workspace_bytes5);
  CUDA_CALL(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      workspace5, workspace_bytes5, _unique_frequency, _unique_frequency,
      _unique_dst, _unique_dst, _num_unique, num_input_node, num_edge_prefix,
      num_edge_prefix + 1, 0, sizeof(IdType) * 8, cu_stream));
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 8 finish";

  // 9. prefix the number of output edges for each nodes that we get in step 7
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace2, workspace_bytes2,
                                          num_output_prefix, num_output_prefix,
                                          num_input_node + 1, cu_stream));
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 9 finish";

  // 10. copy the result to the output array and set the value of num_output
  compact_output<<<grid2, block2, 0, cu_stream>>>(
      _unique_src, _unique_dst, _unique_frequency, num_input_node, K,
      num_edge_prefix, num_output_prefix, output_src, output_dst, output_data,
      num_output);
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 10 finish";

  // 11. reset data
  reset_node_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid1, block1, 0, cu_stream>>>(device_table, _node_list, _num_node);
  Device::Get(_ctx)->StreamSync(_ctx, stream);

  reset_edge_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid3, block3, 0, cu_stream>>>(device_table, _unique_src, _unique_dst,
                                        _num_unique);
  Device::Get(_ctx)->StreamSync(_ctx, stream);

  _num_node = 0;
  _num_unique = 0;

  device->FreeWorkspace(_ctx, workspace5);
  device->FreeWorkspace(_ctx, num_output_prefix);
  device->FreeWorkspace(_ctx, num_edge_prefix);
  device->FreeWorkspace(_ctx, workspace4);
  device->FreeWorkspace(_ctx, workspace3);
  device->FreeWorkspace(_ctx, device_num_unique);
  device->FreeWorkspace(_ctx, num_unique_prefix);
  device->FreeWorkspace(_ctx, workspace2);
  device->FreeWorkspace(_ctx, workspace1);
  device->FreeWorkspace(_ctx, workspace0);
}

}  // namespace cuda
}  // namespace common
}  // namespace samgraph