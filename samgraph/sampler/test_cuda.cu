
#include <hip/hip_runtime.h>
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}
