#include "hip/hip_runtime.h"
#include <>

struct KeyValue
{
    uint32_t key;
    uint32_t value;
};

void gpu_hashtable_insert(KeyValue* hashtable, uint32_t key, uint32_t value)
{
    uint32_t slot = hash(key);

    while (true)
    {
        uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
        if (prev == kEmpty || prev == key)
        {
            hashtable[slot].value = value;
            break;
        }
        slot = (slot + 1) & (kHashTableCapacity-1);
    }
}